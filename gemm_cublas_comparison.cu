#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <iostream>

const int N = 10240;

void checkCudaError(hipError_t err) {
    if (err != hipSuccess) {
        std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
        exit(EXIT_FAILURE);
    }
}

void checkCublasError(hipblasStatus_t err) {
    if (err != HIPBLAS_STATUS_SUCCESS) {
        std::cerr << "cuBLAS Error" << std::endl;
        exit(EXIT_FAILURE);
    }
}

int main() {
    float *d_A, *d_B, *d_C;
    hipblasHandle_t handle;

    // Allocate memory on the device
    checkCudaError(hipMalloc(&d_A, N * N * sizeof(float)));
    checkCudaError(hipMalloc(&d_B, N * N * sizeof(float)));
    checkCudaError(hipMalloc(&d_C, N * N * sizeof(float)));

    // Create cuBLAS handle
    checkCublasError(hipblasCreate(&handle));

    // Initialize matrices (fill with some values, here we skip actual initialization for brevity)

    float alpha = 1.0f;
    float beta = 0.0f;

    // Timing cuBLAS GEMM
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    checkCublasError(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha, d_A, N, d_B, N, &beta, d_C, N));
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    std::cout << "cuBLAS GEMM took " << milliseconds << " ms" << std::endl;

    // Clean up
    hipblasDestroy(handle);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
