#include <stdio.h>
#include <hip/hip_runtime.h>

#define TILE_SIZE 32 

__global__ void matrixMulSharedMemory(float *A, float *B, float *C, int N) {
    __shared__ float tileA[TILE_SIZE][TILE_SIZE];
    __shared__ float tileB[TILE_SIZE][TILE_SIZE];

    int row = blockIdx.y * TILE_SIZE + threadIdx.y;
    int col = blockIdx.x * TILE_SIZE + threadIdx.x;
    float value = 0;

    for (int i = 0; i < (N + TILE_SIZE - 1) / TILE_SIZE; i++) {
        if (row < N && i * TILE_SIZE + threadIdx.x < N) {
            tileA[threadIdx.y][threadIdx.x] = A[row * N + i * TILE_SIZE + threadIdx.x];
        } else {
            tileA[threadIdx.y][threadIdx.x] = 0.0f;
        }

        if (col < N && i * TILE_SIZE + threadIdx.y < N) {
            tileB[threadIdx.y][threadIdx.x] = B[(i * TILE_SIZE + threadIdx.y) * N + col];
        } else {
            tileB[threadIdx.y][threadIdx.x] = 0.0f;
        }

        __syncthreads();

        for (int j = 0; j < TILE_SIZE; j++) {
            value += tileA[threadIdx.y][j] * tileB[j][threadIdx.x];
        }

        __syncthreads();
    }

    if (row < N && col < N) {
        C[row * N + col] = value;
    }
}

int main() {
    int N = 10240;
    size_t size = N * N * sizeof(float);

    // Allocate memory on the host
    float *h_A = (float*)malloc(size);
    float *h_B = (float*)malloc(size);
    float *h_C = (float*)malloc(size);

    // Initialize matrices A and B
    for (int i = 0; i < N * N; i++) {
        h_A[i] = 1.0f;  // Example initialization
        h_B[i] = 1.0f;  // Example initialization
    }

    // Allocate memory on the device
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    // Copy data to the device
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Define the grid and block dimensions
    dim3 dimBlock(TILE_SIZE, TILE_SIZE);
    dim3 dimGrid((N + TILE_SIZE - 1) / TILE_SIZE, (N + TILE_SIZE - 1) / TILE_SIZE);

    // Create CUDA events to measure time
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Record start time
    hipEventRecord(start);

    // Launch the optimized kernel
    matrixMulSharedMemory<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, N);

    // Record end time
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    // Calculate elapsed time
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    printf("Time for the kernel: %f ms\n", milliseconds);

    // Copy the result back to the host
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}
