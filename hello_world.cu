#include <stdio.h>
#include <hip/hip_runtime.h>

// Kernel function to print "Hello, World!" from the GPU
__global__ void helloWorldKernel() {
    printf("Hello, World from GPU!\n");
}

void checkCudaError(hipError_t err, const char* msg) {
    if (err != hipSuccess) {
        fprintf(stderr, "Error: %s: %s\n", msg, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

int main() {
    helloWorldKernel<<<1, 1>>>();

    hipError_t err = hipGetLastError();
    checkCudaError(err, "Kernel launch failed");

    err = hipDeviceSynchronize();
    checkCudaError(err, "Device synchronization failed");

    return 0;
}




